#include "testBase.h"

__host__ __device__ int steps() {return 3;}

__host__ __device__ 
int stepSize(int k) {
  return 1;
}

__device__ inline
VertexID next(int step, const VertexID transit, const VertexID sample, 
              const float max_weight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, hiprandState* state)
{
  EdgePos_t id = RandNumGen::rand_int(state, numEdges);
  return transitEdges[id];
}

template<int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool DECREASE_GM_LOADS>
__device__ inline
VertexID nextCached(int step, const VertexID transit, const VertexID sample, 
              const float max_weight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, 
              hiprandState* state, VertexID_t* cachedEdges, float* cachedWeights,
              bool* globalLoadBV)
{
  EdgePos_t x = RandNumGen::rand_int(state, numEdges);
  if (CACHE_EDGES)
    return cacheAndGet<CACHE_SIZE, DECREASE_GM_LOADS>(x, transitEdges, cachedEdges, globalLoadBV);
  else 
    return transitEdges[x];
}

__host__ int samplingType()
{
  return SamplingType::Individual;
}

//nvprof bin/test_rw_10.2_x86_64 by-pass --graph-file=/mnt/homes/abhinav/GPUesque-for-eval/input/reddit_sampled_matrix --walks-per-node=1 --walk-length=10 --walk-mode=0

//APP_TEST(UniformRandWalk, CiteseerTP, GRAPH_PATH"/citeseer-weighted.graph", 10, false, "TransitParallel") 
// APP_TEST(UniformRandWalk, CiteseerSP, GRAPH_PATH"/citeseer-weighted.graph", 10, false, "SampleParallel") 
// APP_TEST(UniformRandWalk, MicoTP, GRAPH_PATH"/micro-weighted.graph", 10, false, "TransitParallel")
// APP_TEST(UniformRandWalk, MicoSP, GRAPH_PATH"/micro-weighted.graph", 10, false, "SampleParallel") 
// APP_TEST(UniformRandWalk, PpiTP, GRAPH_PATH"/ppi_sampled_matrix", 10, false, "TransitParallel")
// APP_TEST(UniformRandWalk, PpiSP, GRAPH_PATH"/ppi_sampled_matrix", 10, false, "SampleParallel")
// APP_TEST(UniformRandWalk, RedditTP, GRAPH_PATH"/reddit_sampled_matrix", 10, false, "TransitParallel", false)
//APP_TEST(UniformRandWalk, RedditLB, GRAPH_PATH"/reddit_sampled_matrix", 1, true, "TransitParallel", true)
// APP_TEST(UniformRandWalk, RedditSP, GRAPH_PATH"/reddit_sampled_matrix", 10, false, "SampleParallel")
// APP_TEST(UniformRandWalk, OrkutTP, GRAPH_PATH"/com-orkut-weighted.graph", 10, false, "TransitParallel", false)
// APP_TEST(UniformRandWalk, OrkutLB, GRAPH_PATH"/com-orkut-weighted.graph", 10, false, "TransitParallel", true)
// APP_TEST(UniformRandWalk, OrkutSP, GRAPH_PATH"/com-orkut-weighted.graph", 10, false, "SampleParallel", false)
// APP_TEST(UniformRandWalk, LiveJournalTP, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", 10, false, "TransitParallel", false)
APP_TEST(UniformRandWalk, LiveJournalLB, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", 1, true, "TransitParallel", true)
// APP_TEST(UniformRandWalk, LiveJournalSP, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", 10, false, "SampleParallel", false)