#include "hip/hip_runtime.h"
#include "testBase.h"

#include <stdlib.h>
#include "../apps/multiRW/multiRW.cu"

#define RUNS 1
#define CHECK_RESULTS false
#define VERTICES_PER_SAMPLE 0
#include "../check_results.cu"

template<class SampleType, typename App>
bool checkMultiRWResult(NextDoorData<SampleType, App>& nextDoorData)
{
  //Check result by traversing all sampled neighbors and making
  //sure that if neighbors at kth-hop is an adjacent vertex of one
  //of the k-1th hop neighbors.
  CSR* csr = nextDoorData.csr;
  auto& initialSamples = nextDoorData.initialContents;
  auto finalSampleSize = getFinalSampleSize<MultiRWApp>();
  auto& finalSamples = nextDoorData.hFinalSamples;
  auto INVALID_VERTEX = nextDoorData.INVALID_VERTEX;
  auto& samples = nextDoorData.samples;
  int maxSteps = 4;

  //First create the adjacency matrix.
  std::cout << "checking results" << std::endl;
  AdjMatrix adj_matrix;

  csrToAdjMatrix(csr, adj_matrix);

  //Now check the correctness
  size_t numNeighborsToSampleAtStep = 0;
  
  for (int step = 0; step < min(maxSteps, MultiRWApp().steps()); step++) {
    if (step == 0) { 
      bool foundError = false;
      #pragma omp parallel for shared(foundError)
      for (size_t s = 0; s < finalSamples.size(); s += finalSampleSize) {
        std::unordered_set<VertexID_t> uniqueNeighbors;
        // printf("omp_get_num_threads() %d\n", omp_get_num_threads());
        const size_t sampleId = s/finalSampleSize;
        size_t contentsLength = 0;
        if (MultiRWApp().stepSize(step) != ALL_NEIGHBORS) {
          for (size_t v = s + numNeighborsToSampleAtStep; v < s + MultiRWApp().stepSize(step); v++) {
            VertexID_t transit = finalSamples[v];
            uniqueNeighbors.insert(transit);
            contentsLength += (int)(transit != INVALID_VERTEX);

            bool foundInRootVertices = false;
            for (int ii = 0; ii < NUM_ROOT_VERTICES; ii++) {
              VertexID_t rv = samples[sampleId].rootVertices[ii];
              if (adj_matrix[rv].count(transit) == 1) {
                foundInRootVertices = true;
                break;
              }
            }

            if (!foundError && transit != INVALID_VERTEX &&
                !foundInRootVertices) {
              printf("%s:%d Invalid '%d' in Sample '%ld' at Step '%d'\n", __FILE__, __LINE__, transit, sampleId, step);
              foundError = true;
            }
          }

          if (!foundError && contentsLength == 0) {
            printf("Step %d: '%ld' vertices sampled for sample '%ld' but sum of edges of all vertices in sample is '%ld'\n", 
                    step, contentsLength, sampleId, adj_matrix[samples[sampleId].rootVertices[0]].size());
            foundError = true;
          }
        } 
      }

      if (foundError) return false;
    } else {
      bool foundError = false;
      #pragma omp parallel for shared(foundError)
      for (size_t s = 0; s < finalSamples.size(); s += finalSampleSize) {
        const size_t sampleId = s/finalSampleSize;
        size_t contentsLength = 0;
        size_t sumEdgesOfNeighborsAtPrevStep = 0;
        const VertexID_t initialVal = samples[sampleId].rootVertices[0];

        for (size_t v = s + numNeighborsSampledAtStep<MultiRWApp>(step-2); v < s + numNeighborsSampledAtStep<MultiRWApp>(step-1); v++) {
          sumEdgesOfNeighborsAtPrevStep +=  adj_matrix[finalSamples[v]].size();
        }
        
        // if (sampleId == 48) {
        //   printf("step %d start %d end %d\n", step, numNeighborsSampledAtStep(step-1),
        //          ((step == steps() - 1) ? finalSampleSize : numNeighborsSampledAtStep(step)));
        // }
        for (size_t v = s + numNeighborsSampledAtStep<MultiRWApp>(step-1); 
             v < s + ((step == MultiRWApp().steps() - 1) ? finalSampleSize : numNeighborsSampledAtStep<MultiRWApp>(step)); v++) {
          VertexID_t transit = finalSamples[v];
          contentsLength += (int)(transit != INVALID_VERTEX);
          
          bool foundInRootVertices = false;
          for (auto rv : samples[sampleId].rootVertices) {
            if (adj_matrix[rv].count(transit) == 1) {
              foundInRootVertices = true;
              break;
            }
          }

          if (transit != INVALID_VERTEX) {
            if (!foundError && !foundInRootVertices) {
              printf("%s:%d Invalid '%d' in Sample '%ld' at Step '%d'\n", __FILE__, __LINE__, transit, sampleId, step);
              std::cout << "Contents of sample : [";
              for (size_t v2 = s; v2 < s + finalSampleSize; v2++) {
                std::cout << finalSamples[v2] << ", ";
              }
              std::cout << "]" << std::endl;
              foundError = true;
            }
          }
        }

        if (!foundError && contentsLength == 0 && sumEdgesOfNeighborsAtPrevStep > 0) {
          printf("Step %d: '%ld' vertices sampled for sample '%ld' but sum of edges of all vertices in sample is '%ld'\n", 
                  step, contentsLength, sampleId, sumEdgesOfNeighborsAtPrevStep);
          std::cout << "Contents of sample : [";
          for (size_t v2 = s; v2 < s + finalSampleSize; v2++) {
            std::cout << finalSamples[v2] << ", ";
          }
          std::cout << "]" << std::endl;
          foundError = true;
        }
      }

      if (foundError) return false;
    }

    numNeighborsToSampleAtStep = stepSizeAtStep<MultiRWApp>(step);
  }

  return true;
}

APP_TEST_BINARY(MultiRWSample, MultiRW, MultiRWApp, LiveJournalLB, LJ1_PATH, RUNS, CHECK_RESULTS, 
                checkMultiRWResult, "TransitParallel", true)
APP_TEST_BINARY(MultiRWSample, MultiRW, MultiRWApp, OrkutLB, ORKUT_PATH, RUNS, CHECK_RESULTS, 
                checkMultiRWResult, "TransitParallel", true)
APP_TEST_BINARY(MultiRWSample, MultiRW, MultiRWApp, PatentsLB, PATENTS_PATH, RUNS, CHECK_RESULTS, 
                checkMultiRWResult, "TransitParallel", true)
APP_TEST_BINARY(MultiRWSample, MultiRW, MultiRWApp, RedditLB, REDDIT_PATH, RUNS, CHECK_RESULTS, 
                checkMultiRWResult, "TransitParallel", true)
APP_TEST_BINARY(MultiRWSample, MultiRW, MultiRWApp, PPILB, PPI_PATH, RUNS, CHECK_RESULTS, 
                checkMultiRWResult, "TransitParallel", true)
